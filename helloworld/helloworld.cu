#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "pfc_cuda_device_info.h"
#include "pfc_cuda_memory.h"
#include "pfc_random.h"
#include "pfc_timed_run.h"
#include <iostream>
#include <string>


using namespace std::literals;

__constant__ auto const g_block_size = 128;
__constant__ auto const g_points = 50000;

dim3 grid_size(dim3 const & block, int3 const & size) {

	dim3 s;
	s.x = (size.x + block.x - 1) / block.x;
	s.y = (size.y + block.y - 1) / block.y;
	s.z = (size.z + block.z - 1) / block.z;

	return s;


}



auto const g_grid_size = grid_size(
	g_block_size, { g_points , 1, 1 }
);


__host__  __device__   __forceinline__
float norm(float3 const & p, float3 const & q) {
	auto x = p.x - q.x;
	auto y = p.y - q.y;
	auto z = p.z - q.z;

	return x*x + y*y + z*z;

}

__device__ int global_thread_idx_x() {

	return blockIdx.x * blockDim.x + threadIdx.x;


}




__host__ __device__ __forceinline__
int find_closest(float3 * p_points, float3 const * p_point) {
	int index = -1;
	float min_so_far = FLT_MAX;

	for (int to = 0; to < g_points; ++p_points, ++to) {
		if (p_points != p_point) {
			auto const dist = norm(*p_point, *p_points);

			if (dist < min_so_far) {
				min_so_far = dist; index = to;

			}

		}

	}

	return index;

}


void generate_points(float3 * const hp_points)
{




	for (size_t i = 0; i < g_points; i++)
	{
		hp_points[i].x = pfc::get_random_uniform(0, 5000)*1.0f;
		hp_points[i].y = pfc::get_random_uniform(0, 5000)*1.0f;
		hp_points[i].z = pfc::get_random_uniform(0, 5000)*1.0f;
	}
}


__global__ void find_all_closest_GPU(
	float3 * const dp_points, int * const dp_indices
) {
	auto const from = global_thread_idx_x();

	if (from < g_points)
	{
		dp_indices[from] = find_closest(dp_points, dp_points + from);
	}



}

void find_all_closest_CPU(
	float3 * const hp_points, int * const hp_indices_h
) {


	for (size_t i = 0; i < g_points - 1; i++)
	{
		hp_indices_h[i] = find_closest(hp_points, hp_points + i);
	}


}










void allocate_memory(
	int * & hp_indices_d, int * & hp_indices_h,
	float3 * & hp_points, int * & dp_indices,
	float3 * & dp_points
) {

	hp_indices_d = new int[g_points] {};
	hp_indices_h = new int[g_points] {};
	hp_points = new float3[g_points]{};


	dp_indices = PFC_CUDA_MALLOC(int, g_points);
	dp_points = PFC_CUDA_MALLOC(float3, g_points);

}

void free_memory(
	int * & hp_indices_d, int * & hp_indices_h,
	float3 * & hp_points, int * & dp_indices,
	float3 * & dp_points
) {

	PFC_CUDA_FREE(dp_points);
	PFC_CUDA_FREE(dp_indices);

	delete[] hp_points; hp_points = nullptr;
	delete[] hp_indices_h; hp_indices_h = nullptr;
	delete[] hp_indices_d; hp_indices_d = nullptr;




}


/*
__global__ void copy_string_kernel(char * dp_dest, char * dp_src, int size) {
	int const t = blockIdx.x * blockDim.x + threadIdx.x;

	if (t < size)
	{
		dp_dest[t] = dp_src[t];
	}

}
*/


int main() {

	try {
		int count = 0;
		PFC_CUDA_CHECK(hipGetDeviceCount(&count));
		if (count > 0)
		{
			PFC_CUDA_CHECK(hipSetDevice(0));

			auto deviceinfo = pfc::cuda::get_device_info();
			auto deviceprops = pfc::cuda::get_device_props();

			std::cout << "name: " << deviceprops.name << "\ncc: " << deviceinfo.cc_major << "." << deviceinfo.cc_minor << " \narch: " << deviceinfo.uarch << std::endl;


			int *  hp_indices_d = nullptr;
			int *  hp_indices_h = nullptr;
			float3 *  hp_points = nullptr;
			int * dp_indices = nullptr;
			float3 *  dp_points = nullptr;




			allocate_memory(hp_indices_d, hp_indices_h, hp_points, dp_indices, dp_points);

			generate_points(hp_points);

			std::cout << "CPU" << std::endl;


			auto const duration_cpu = pfc::timed_run([&] {

				find_all_closest_CPU(hp_points, hp_indices_h);
			});

			auto cpu_time = std::chrono::duration_cast<std::chrono::milliseconds>(duration_cpu).count();
			


			std::cout << "GPU" << std::endl;
			auto const duration_gpu = pfc::timed_run([&] {

				PFC_CUDA_MEMCPY(dp_points, hp_points, g_points, hipMemcpyHostToDevice);

				//int const big = (g_points + g_block_size + -1) / g_block_size;


				find_all_closest_GPU << <g_block_size,g_grid_size >> > (dp_points, dp_indices);

				//copy_string_kernel << <big, tib >> > (dp_dest, dp_src, size);

				
				PFC_CUDA_MEMCPY(hp_indices_d, dp_indices, g_points, hipMemcpyDeviceToHost);
				PFC_CUDA_CHECK(hipDeviceSynchronize());
				PFC_CUDA_CHECK(hipGetLastError());

			});

			auto gpu_time =  std::chrono::duration_cast<std::chrono::milliseconds>(duration_gpu).count();

			std::cout << "CPU Time: " << cpu_time << " GPU Time: " << gpu_time << " Speedup: " << (cpu_time*1.0f) / (gpu_time*1.0f) << std::endl;
			

		

			free_memory(hp_indices_d, hp_indices_h, hp_points, dp_indices, dp_points);

		}



	}
	catch (std::exception const & x) {
		std::cerr << "ERROR: " << x.what() << std::endl;

	}

	hipDeviceReset();

}