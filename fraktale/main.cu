#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "pfc_cuda_device_info.h"
#include "pfc_cuda_memory.h"
#include "pfc_random.h"
#include "pfc_timed_run.h"
#include <iostream>
#include <string>
#include <map>
#include "pfc_complex.h"
#include "pfc_rgb_from_wavelength.h"
#include "pfc_bitmap.h"
#include <Windows.h>


typedef std::vector<pfc::bitmap::pixel_t> bmpdata;
typedef std::pair <pfc::complex<>, pfc::complex<>> area;
typedef std::vector<area> points;
typedef std::pair<int, int> picturesize;
typedef std::map<int, bmpdata> datamap;

using namespace std::literals;

__constant__ auto const g_block_size = 128;
__constant__ auto const g_x = 5000;
__constant__ auto const g_y = 5000;
__constant__ auto const d = 4;
__constant__ auto const it = 100;
__constant__ area a = std::pair <pfc::complex<>, pfc::complex<>>((-2, 2), (2, 2));
__constant__ picturesize psize = std::pair<int, int>(g_x, g_y);











dim3 grid_size(dim3 const & block, int3 const & size) {

	dim3 s;
	s.x = (size.x + block.x - 1) / block.x;
	s.y = (size.y + block.y - 1) / block.y;
	s.z = (size.z + block.z - 1) / block.z;

	return s;


}



auto const g_grid_size = grid_size(
	g_block_size, { g_x , g_y, 1 }
);


__host__  __device__   __forceinline__
float norm(float3 const & p, float3 const & q) {
	auto x = p.x - q.x;
	auto y = p.y - q.y;
	auto z = p.z - q.z;

	return x*x + y*y + z*z;

}

__device__ int global_thread_idx_x() {

	return blockIdx.x * blockDim.x + threadIdx.x;


}
__device__ int global_thread_idx_y() {

	return blockIdx.y * blockDim.y + threadIdx.y;


}


void color_map(int it, pfc::bitmap::pixel_t *  hp_colors) {

	bmpdata v;

	double w = 1.0 / it;
	double x = 0;
	for (size_t i = 0; i < it + 1; i++)
	{
		x += w;

		pfc::bitmap::pixel_t t;

		pfc::rgb_from_wavelength(t, x);

		v.push_back(t);

	}

	hp_colors = v.data();


	

};


__device__ __forceinline__
int point(pfc::complex<> c, int it, size_t d) {

	pfc::complex<> start = 0;

	pfc::complex<> e = start;

	int i = 0;

	do
	{
		e = pfc::square(e) + c;
	} while ((++i < it) && pfc::norm(e) < d);


	return i;
};

__device__ __forceinline__
pfc::complex<> mapping(picturesize p, int x, int y, area a) {

	double isize = abs(a.first.imag - a.second.imag);
	double rsize = abs(a.first.real - a.second.real);

	double ix = x*(rsize / p.first) + a.first.real;
	double iy = y*(isize / p.second) + a.first.imag;


	return pfc::complex<>(ix, iy);




};


__global__ void fraktal_GPU(pfc::bitmap::pixel_t * const dp_colors, pfc::bitmap::pixel_t * const dp_picture) {

	int x = global_thread_idx_x();
	int y = global_thread_idx_y();


	if (x < g_x && y < g_y) {

		int p = point(mapping(psize, x, y, a), it, d);

		pfc::bitmap::pixel_t rgb = dp_colors[p];
		int val = x * g_x + y;

		dp_picture[val] = rgb;

	}


}


void allocate_memory(
	pfc::bitmap::pixel_t * & hp_colors, pfc::bitmap::pixel_t * & hp_picture,
	pfc::bitmap::pixel_t * & dp_colors,
	pfc::bitmap::pixel_t *& dp_picture){

	hp_colors = new pfc::bitmap::pixel_t[it] {};
	hp_picture = new  pfc::bitmap::pixel_t[g_x*g_y] {};
	
	

	dp_colors = PFC_CUDA_MALLOC(pfc::bitmap::pixel_t, it);
	dp_picture = PFC_CUDA_MALLOC(pfc::bitmap::pixel_t, g_x*g_y);

	double m = (((2.0 * sizeof(int)*it + sizeof(pfc::bitmap::pixel_t))*g_x*g_y) / 1024.0) / 1024.0;
	std::cout << "Memory allocated : " << m <<" Mib"<< std::endl;
}

void free_memory(
	pfc::bitmap::pixel_t * & hp_colors, pfc::bitmap::pixel_t * & hp_picture,
	pfc::bitmap::pixel_t * & dp_colors,
	pfc::bitmap::pixel_t *& dp_picture
) {

	PFC_CUDA_FREE(dp_colors);
	PFC_CUDA_FREE(dp_picture);

	delete[] hp_colors; hp_colors = nullptr;
	delete[] hp_picture; hp_picture = nullptr;
	double m = (((2.0 * sizeof(pfc::bitmap::pixel_t)*it + 2.0 * (sizeof(pfc::bitmap::pixel_t))*g_x*g_y) / 1024.0) / 1024.0;
	std::cout << "Memory freed : " << m << " Mib" << std::endl;


}



int main() {

	try {
		int count = 0;
		PFC_CUDA_CHECK(hipGetDeviceCount(&count));
		if (count > 0)
		{
			PFC_CUDA_CHECK(hipSetDevice(0));

			auto deviceinfo = pfc::cuda::get_device_info();
			auto deviceprops = pfc::cuda::get_device_props();

			std::cout << "Name: " << deviceprops.name << "\ncc: " << deviceinfo.cc_major << "." << deviceinfo.cc_minor << " \nArch: " << deviceinfo.uarch << std::endl;

			std::cout << "Points: " << g_x*g_y << std::endl;
			std::cout << "Threads: " << g_block_size << std::endl;
			std::cout << "Blocks: " << g_grid_size.x << std::endl;



			pfc::bitmap::pixel_t *  hp_colors = nullptr;
			pfc::bitmap::pixel_t *  hp_picture = nullptr;
			pfc::bitmap::pixel_t * dp_colors = nullptr;
			pfc::bitmap::pixel_t *  dp_picture = nullptr;
		

			color_map(it,hp_colors);

			std::cout << "allocating memory:" << std::endl;
			allocate_memory(hp_colors, hp_picture , dp_colors, dp_picture);
		


			std::cout << "GPU" << std::endl;
			std::cout << "coping to device ("<< sizeof(int)*it/1024.0/1024.0 <<" Mib)" << std::endl;
			auto const duration_gpu = pfc::timed_run([&] {

				
				PFC_CUDA_MEMCPY(dp_colors, hp_colors, it, hipMemcpyHostToDevice);

				fraktal_GPU <<<g_grid_size,g_block_size >>> (dp_colors, dp_picture);

				
				PFC_CUDA_MEMCPY(hp_picture, dp_picture, g_x*g_y, hipMemcpyDeviceToHost);
				PFC_CUDA_CHECK(hipDeviceSynchronize());
				PFC_CUDA_CHECK(hipGetLastError());

			});

			std::cout << "coping to host (" << sizeof(pfc::bitmap::pixel_t)*g_x*g_y / 1024.0 / 1024.0 << " Mib)" << std::endl;

			auto gpu_time =  std::chrono::duration_cast<std::chrono::milliseconds>(duration_gpu).count();

		
			
/*
			for (size_t i = 0; i < g_points; i++)
			{
				std::cout << hp_indices_d[i] << "-" << hp_indices_h[i] << std::endl;
			}

			*/
	free_memory(hp_colors, hp_picture, dp_colors, dp_picture);
	std::cout <<" GPU Time: " << gpu_time  << std::endl;

	pfc::bitmap bmp;

	bmp.get_pixels = hp_picture;

	bmp.to_file("fraktal.bmp");

		}



	}
	catch (std::exception const & x) {
		std::cerr << "ERROR: " << x.what() << std::endl;

	}

	hipDeviceReset();

}