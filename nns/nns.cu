#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include "pfc_cuda_device_info.h"
#include "pfc_cuda_memory.h"
#include "pfc_random.h"
#include "pfc_timed_run.h"
#include <iostream>
#include <string>


using namespace std::literals;

__constant__ auto const g_block_size = 128;
__constant__ auto const g_points = 60000;







dim3 grid_size(dim3 const & block, int3 const & size) {

	dim3 s;
	s.x = (size.x + block.x - 1) / block.x;
	s.y = (size.y + block.y - 1) / block.y;
	s.z = (size.z + block.z - 1) / block.z;

	return s;


}



auto const g_grid_size = grid_size(
	g_block_size, { g_points , 1, 1 }
);


__host__  __device__   __forceinline__
float norm(float3 const & p, float3 const & q) {
	auto x = p.x - q.x;
	auto y = p.y - q.y;
	auto z = p.z - q.z;

	return x*x + y*y + z*z;

}

__device__ int global_thread_idx_x() {

	return blockIdx.x * blockDim.x + threadIdx.x;


}




__host__ __device__ __forceinline__
int find_closest(float3 * p_points, float3 const * p_point) {
	int index = -1;
	float min_so_far = FLT_MAX;

	for (int to = 0; to < g_points; ++p_points, ++to) {
		if (p_points != p_point) {
			auto const dist = norm(*p_point, *p_points);

			if (dist < min_so_far) {
				min_so_far = dist; index = to;

			}

		}

	}

	return index;

}


void generate_points(float3 * const hp_points)
{




	for (size_t i = 0; i < g_points; i++)
	{
		hp_points[i].x = pfc::get_random_uniform(0, 5000)*1.0f;
		hp_points[i].y = pfc::get_random_uniform(0, 5000)*1.0f;
		hp_points[i].z = pfc::get_random_uniform(0, 5000)*1.0f;
	}
}


__global__ void find_all_closest_GPU(
	float3 * const dp_points, int * const dp_indices
) {
	auto const from = global_thread_idx_x();

	if (from < g_points)
	{
		dp_indices[from] = find_closest(dp_points, dp_points + from);
	}



}

std::vector<std::pair<int, int>> pointrange(int num) {

	std::vector<std::pair<int, int>> vec;
	int range = g_points / num;
		std::pair<int,int> r;
		r.first = 0;
		r.second = range;

	for (size_t i = 0; i < num; i++)
	{
		vec.push_back(std::pair<int, int> (r.first,r.second-1) );
		r.first += range;
		r.second += range;
		
	}
	return vec;
}

void find_all_closest_CPU_SC(
	float3 * const hp_points, int * const hp_indices_h
) {
	for (size_t j = 0; j <g_points; j++)
	{
		hp_indices_h[j] = find_closest(hp_points, hp_points + j);
	}

}






void find_all_closest_CPU_MC(
	float3 * const hp_points, int * const hp_indices_h,std::vector<std::pair<int,int>> parts
) {

	

	std::vector<std::thread> group;


	for (int i = 0; i < parts.size(); i++)
	{

	

			group.emplace_back([&parts,i,hp_indices_h,hp_points]() {

			for (size_t j =parts.at(i).first ; j <parts.at(i).second; j++)
	{
		hp_indices_h[j] = find_closest(hp_points, hp_points + j);
	}

			});
		
	}

	for (auto &t : group)
	{
		t.join();
	}
}










void allocate_memory(
	int * & hp_indices_d, int * & hp_indices_h,
	float3 * & hp_points, int * & dp_indices,
	float3 * & dp_points
) {

	hp_indices_d = new int[g_points] {};
	hp_indices_h = new int[g_points] {};
	hp_points = new float3[g_points]{};
	

	dp_indices = PFC_CUDA_MALLOC(int, g_points);
	dp_points = PFC_CUDA_MALLOC(float3, g_points);

	double m = (((3.0 * sizeof(int) + 2.0 * sizeof(float3))*g_points) / 1024.0) / 1024.0;
	std::cout << "Memory allocated : " << m <<" Mib"<< std::endl;
}

void free_memory(
	int * & hp_indices_d, int * & hp_indices_h,
	float3 * & hp_points, int * & dp_indices,
	float3 * & dp_points
) {

	PFC_CUDA_FREE(dp_points);
	PFC_CUDA_FREE(dp_indices);

	delete[] hp_points; hp_points = nullptr;
	delete[] hp_indices_h; hp_indices_h = nullptr;
	delete[] hp_indices_d; hp_indices_d = nullptr;
	double m = (((3.0 * sizeof(int) + 2.0 * sizeof(float3))*g_points) / 1024.0) / 1024.0;
	std::cout << "Memory freed : " << m << " Mib" << std::endl;


}



int main() {

	try {
		int count = 0;
		PFC_CUDA_CHECK(hipGetDeviceCount(&count));
		if (count > 0)
		{
			PFC_CUDA_CHECK(hipSetDevice(0));

			auto deviceinfo = pfc::cuda::get_device_info();
			auto deviceprops = pfc::cuda::get_device_props();

			std::cout << "Name: " << deviceprops.name << "\ncc: " << deviceinfo.cc_major << "." << deviceinfo.cc_minor << " \nArch: " << deviceinfo.uarch << std::endl;

			std::cout << "Points: " << g_points << std::endl;
			std::cout << "Threads: " << g_block_size << std::endl;
			std::cout << "Blocks: " << g_grid_size.x << std::endl;



			int *  hp_indices_d = nullptr;
			int *  hp_indices_h = nullptr;
			float3 *  hp_points = nullptr;
			int * dp_indices = nullptr;
			float3 *  dp_points = nullptr;



			std::cout << "allocating memory:" << std::endl;
			allocate_memory(hp_indices_d, hp_indices_h, hp_points, dp_indices, dp_points);
			
			
			std::cout << "generating points" << std::endl;
			generate_points(hp_points);

			std::cout << "CPU" << std::endl;


			auto const duration_cpu = pfc::timed_run([&] {

				find_all_closest_CPU_SC(hp_points, hp_indices_h);
			});

			auto cpu_time = std::chrono::duration_cast<std::chrono::milliseconds>(duration_cpu).count();
			


			std::cout << "GPU" << std::endl;
			std::cout << "coping to device ("<< sizeof(float3)*g_points/1024.0/1024.0 <<" Mib)" << std::endl;
			auto const duration_gpu = pfc::timed_run([&] {

				
				PFC_CUDA_MEMCPY(dp_points, hp_points, g_points, hipMemcpyHostToDevice);

				find_all_closest_GPU << <g_grid_size,g_block_size >> > (dp_points, dp_indices);

				
				PFC_CUDA_MEMCPY(hp_indices_d, dp_indices, g_points, hipMemcpyDeviceToHost);
				PFC_CUDA_CHECK(hipDeviceSynchronize());
				PFC_CUDA_CHECK(hipGetLastError());

			});

			std::cout << "coping to host (" << sizeof(float3)*g_points / 1024.0 / 1024.0 << " Mib)" << std::endl;

			auto gpu_time =  std::chrono::duration_cast<std::chrono::milliseconds>(duration_gpu).count();

		
			
/*
			for (size_t i = 0; i < g_points; i++)
			{
				std::cout << hp_indices_d[i] << "-" << hp_indices_h[i] << std::endl;
			}

			*/
	free_memory(hp_indices_d, hp_indices_h, hp_points, dp_indices, dp_points);
	std::cout << "CPU Time: " << cpu_time << " GPU Time: " << gpu_time << " Speedup: " << (cpu_time*1.0f) / (gpu_time*1.0f) << std::endl;
		}



	}
	catch (std::exception const & x) {
		std::cerr << "ERROR: " << x.what() << std::endl;

	}

	hipDeviceReset();

}